/***********************************************************
 * A Cuda translation of Aaron's C implementation          *
 *   of Flash Attention; Date: June 2024                   *
 ***********************************************************/


#include <hip/hip_runtime.h>
#include <assert.h>
#include <stdio.h>
#include <stdlib.h>
#include <stdbool.h>
#include <string.h>
#include <math.h>
#include <sys/time.h>

#define GPU_RUNS 50

__host__ int gpuAssert(hipError_t code) {
  if(code != hipSuccess) {
    printf("GPU Error: %s\n", hipGetErrorString(code));
    return -1;
  }
  return 0;
}


__device__ __forceinline__ float atomicMaxFloat (float * addr, float value) {
    float old;
    old = (value >= 0) ? __int_as_float(atomicMax((int *)addr, __float_as_int(value))) :
         __uint_as_float(atomicMin((unsigned int *)addr, __float_as_uint(value)));

    return old;
}

template <const int BN, const int TN, const int mul>
__global__ void __launch_bounds__((BN * BN) / (TN * TN), 1)
s2mm_kernel(int N, int d, const float* Q, const float* K, const float* V, float* O, float* ms) {

  const int num_block_out = BN * BN;
  const int num_thread_out = TN * TN;
  const int num_threads = num_block_out / num_thread_out;
  assert(num_threads == blockDim.x);

  // Shared memory
  extern __shared__ float shared_memory[];
  float* shared_Q = shared_memory;
  float* shared_K = shared_memory + BN * BN * mul;
  float* shared_P = shared_K + BN * (BN+1);
  float* shared_V = shared_K;

  float* maxs = shared_P + BN * BN;   // [BN]
  float* sums = maxs + BN;            // [BN]
  float* es   = sums + BN;            // [BN]
  float* el   = es   + BN;            // [BN]
  float* li   = el   + BN;            // [BN]
  float* mi   = li   + BN;            // [BN]

  // __shared__ float shared_Q[BN * BN * mul];
  // __shared__ float shared_K[BN * BN];
  // __shared__ float shared_P[BN * BN];

  // Registers
  float reg_O[TN * TN * mul] = {0.0};

  const float* ptr_Q;
  const float* ptr_K;
  const float* ptr_V;
  float* ptr_O;

  const int thread_row_Q = threadIdx.x / BN;
  const int thread_col_Q = threadIdx.x % BN;
  const int stride_Q = num_threads / BN;

//  const int thread_row_K = thread_row_Q;
//  const int thread_col_K = thread_col_Q;
//  const int stride_K = stride_Q;

  const int thread_row_V = thread_row_Q;
  const int thread_col_V = thread_col_Q;
  const int stride_V = stride_Q;

  const int thread_row_out = threadIdx.x / (BN / TN);
  const int thread_col_out = threadIdx.x % (BN / TN);

  const int block_row = blockIdx.x;
  const int num_col_blocks = N / BN;

  // initialize mi, li
  for (int t = threadIdx.x; t < BN; t+=num_threads) {
    mi[t]   = -INFINITY;
    li[t]   = 0;
  }


  ptr_Q = Q + block_row * BN * d;
  float *ptr_shared_Q = shared_Q;
  for (int bidx = 0; bidx < d; bidx += BN) {
    for (int offset = 0; offset < BN; offset += stride_Q) {
      ptr_shared_Q[(thread_row_Q + offset) * BN + thread_col_Q] = ptr_Q[(thread_row_Q + offset) * d + thread_col_Q];
    }
    ptr_Q += BN;
    ptr_shared_Q += BN * BN;
  }

  for (int block_col = 0; block_col < num_col_blocks; ++block_col) {
  
    // initialize maxs, sums
    for (int t = threadIdx.x; t < BN; t+=num_threads) {
      sums[t] = 0;
      maxs[t] = -INFINITY;
    }

    ptr_Q = Q + block_row * BN * d;
    ptr_K = K + block_col * BN;
    // ptr_K = K + block_col * BN * d;
    ptr_V = V + block_col * BN * d;
    ptr_O = O + block_row * BN * d;

    float reg_i[TN] = {0.0};
    float reg_j[TN] = {0.0};
    float reg_P[TN * TN] = {0.0};

    ptr_shared_Q = shared_Q;
    for (int bidx = 0; bidx < d; bidx += BN) {
    
      // This works for input K.T, please FIX ME
#if 0
      for (int offset = 0, offset_global = 0; offset < BN; offset += stride_K, offset_global += stride_Q) {
        shared_K[(thread_row_K + offset) * BN + thread_col_K] = ptr_K[(thread_row_K + offset) * N + thread_col_K];
        // shared_K[(thread_row_K + offset) * BN + thread_col_K] = ptr_K[thread_col_Q * d + thread_row_Q + offset_global];
      }
#else
      { // copies from global to shared memory the slice: 
        // K[block_col*BN : block_col*(BN+1)][bidx : bidx + BN]
        for(int tt = 0; tt < TN*TN; tt ++) {
            int row_idx = tt*stride_Q + thread_row_Q;
            // Kj[q*(d+1) + r] = K[glb_ind];
            shared_K[row_idx*(BN+1) + thread_col_Q] = ptr_K[row_idx*N + thread_col_Q];
        }
      }
#endif
      __syncthreads();

      ptr_K += BN;

      for (int idx = 0; idx < BN; ++idx) {

#if 1
        for (int i = 0; i < TN; ++i) {
          reg_i[i] = ptr_shared_Q[(thread_row_out * TN + i) * BN + idx];
        }
        for (int i = 0; i < TN; ++i) {
          //reg_j[i] = shared_K[idx * BN + thread_col_out * TN + i];
          reg_j[i] = shared_K[(thread_col_out * TN + i) * (BN+1) + idx];
        }
        for (int i = 0; i < TN; ++i) {
          for (int j = 0; j < TN; ++j) {
            reg_P[i * TN + j] += reg_i[i] * reg_j[j];
          }
        }
#else
        for (int i = 0; i < TN; ++i) {
          for (int j = 0; j < TN; ++j) {
            reg_P[i * TN + j] += ptr_shared_Q[(thread_row_out * TN + i) * BN + idx] * 
                                 shared_K[(thread_col_out * TN + j) * (BN+1) + idx] ;
          }
        }
#endif
      }

      ptr_shared_Q += BN * BN;
      __syncthreads();
    }

    ///////////////////////////////////
    // reductions of softmax layer
    ///////////////////////////////////

    for (int i = 0; i < TN; i++) {
      float loc_max = reg_P[i * TN + 0];
      for (int j = 1; j < TN; j++) {
        //atomicMaxFloat(&maxs[thread_row_out*TN + i], reg_P[i * TN + j]);
        if (loc_max < reg_P[i * TN + j])
            loc_max = reg_P[i * TN + j];
        //loc_max = max(loc_max, reg_P[i * TN + j]);
      }
      const int ii = thread_row_out*TN + i;
      atomicMaxFloat(&maxs[ii], loc_max);
    }
    __syncthreads();

    #pragma unroll
    for (int i = 0; i < TN; i++) {
      const int ii = thread_row_out*TN + i;
      float row_max = maxs[ii];
      float loc_sum = 0;
      #pragma unroll
      for (int j = 0; j < TN; j++) {
        float pij = reg_P[i * TN + j];
        pij = exp( pij - row_max );
        loc_sum += pij;
        reg_P[i * TN + j] = pij;
      }
      //atomicAdd(&sums[ii], loc_sum); // this is very expensive for some reason; hence we serialize it
      shared_P[thread_col_out*BN + ii] = loc_sum;
    }
    __syncthreads();

    for(int t=threadIdx.x; t < BN; t+=num_threads) {
        float fin_sum = 0;
        #pragma unroll
        for(int j=0; j<BN/TN; j++) {
            fin_sum += shared_P[j*BN + t];
        }
        sums[t] = fin_sum;
    }
    __syncthreads();

    if(threadIdx.x < BN) { // must hold: BN >= TN*TN
      const int ii = threadIdx.x;
      float mi_old = mi[ii];
      float mx = maxs[ii];
      float mi_new = (mi_old > mx) ? mi_old : mx;
      float eij = exp(mx - mi_new);
      float eli = li[ii] * exp(mi_old - mi_new);
      float li_new = eli + sums[ii] * eij;

      mi[ii] = mi_new;
      li[ii] = li_new;
      es[ii] = eij;
      el[ii] = eli;
    }
    __syncthreads();

    //Pij[ii*Bc+jj] = es[ii]*pij;
    for (int i = 0; i < TN; i++) {
      int ii = thread_row_out*TN + i;
      for (int j = 0; j < TN; j++) {
        float pij = reg_P[i * TN + j];
        reg_P[i * TN + j] = es[ii]*pij;
      }
    }

    ///////////////////////////////////
    // end reductions of softmax layer
    ///////////////////////////////////


    for (int i = 0; i < TN; ++i) {
      for (int j = 0; j < TN; ++j) {
        shared_P[(thread_row_out * TN + i) * BN + thread_col_out * TN + j] = reg_P[i * TN + j];
      }
    }

    // here insert the softmax layer

    float* ptr_reg_O = reg_O;
    for (int k = 0; k < mul; k++) {
      for (int offset = 0; offset < BN; offset += stride_V) {
        shared_V[(thread_row_V + offset) * BN + thread_col_V] = ptr_V[(thread_row_V + offset) * d + thread_col_V];
      }
      __syncthreads();

      ptr_V += BN;

      int row_offset = thread_row_out * TN;

      // ptr_reg_O *= el[ii];
      for (int i = 0; i < TN; i++) {
        int ii = row_offset + i;
        for (int j = 0; j < TN; j++) {
            ptr_reg_O[i * TN + j] *= el[ii];
        }
      }

      // calculate per-thread results
      for (int idx = 0; idx < BN; ++idx) {
#if 1
        // block into registers
        for (int i = 0; i < TN; ++i) {
          reg_i[i] = shared_P[(row_offset + i) * BN + idx];
        }
        for (int i = 0; i < TN; ++i) {
          reg_j[i] = shared_V[idx * BN + thread_col_out * TN + i];
        }
        for (int i = 0; i < TN; ++i) {
          for (int j = 0; j < TN; ++j) {
            ptr_reg_O[i * TN + j] += reg_i[i] * reg_j[j];
          }
        }
#else
        for (int i = 0; i < TN; ++i) {
          for (int j = 0; j < TN; ++j) {
            ptr_reg_O[i * TN + j] += shared_P[(row_offset + i) * BN + idx] * //reg_i[i] * 
                                     shared_V[idx * BN + thread_col_out * TN + j]; //reg_j[j];
          }
        }
#endif
      }

      // ptr_reg_O /= li[ii];
      for (int i = 0; i < TN; i++) {
        int ii = row_offset + i;
        for (int j = 0; j < TN; j++) {
            ptr_reg_O[i * TN + j] /= li[ii];
        }
      }

      ptr_reg_O += TN * TN;
      __syncthreads();
    }
    __syncthreads();
  }

  // write out the results
  ptr_O = O + block_row * BN * d;
  float* ptr_reg_O = reg_O;
  for (int bidx = 0; bidx < d; bidx += BN) {
    for (int i = 0; i < TN; ++i) {
      for (int j = 0; j < TN; ++j) {
        ptr_O[(thread_row_out * TN + i) * d + thread_col_out * TN + j] = ptr_reg_O[i * TN + j];
      }
    }
    ptr_O += BN;
    ptr_reg_O += TN * TN;
  }

#if 0
  // For debugging: copy ms back to global memory
  for (int t = threadIdx.x; t < BN; t+=num_threads) {
    int64_t glb_ind = block_row * BN + t;
    ms[glb_ind] = mi[t];
  }
#endif

}

////////////////////////////////////////////////////////////////////////
////////////////////////////////////////////////////////////////////////
////////////////////////////////////////////////////////////////////////

__host__ float
L2(float *x, size_t c)
{
        float sum;

        sum = 0;

        for (size_t i = 0; i < c; i++)
                sum += x[i] * x[i];

        return sqrt(sum);
}

__host__ int
s2mm(float* m_d, float* l_d, float *O_d, float *Q_d, float *K_d, float *V_d, int N, int d)
{
    // int Br, Bc, Tr, Tc;

    // Bc = M / (4 * d);
    // Br = d < Bc ? d : Bc;
    // Tr = N / Br;
    // Tc = N / Bc;

    const int BN = 64; // Br = Bc = Bd = BN
    //const int Bd = BN;
    const int TN = 4;
    const int mul = 2; // d / Bd;

    // // iniatialize m with -INFINITY and l with zero (both have size N)
    // initKer<<<(N+255)/256, 256>>>(m_d, l_d, N);

    // initialize O to zeros
    hipMemset((void**)&O_d, 0, N*d*sizeof(float));

    // setup execution parameters
    // dim3 block(Bc, Br, 1);
    // dim3 grid (Tr,  1, 1);
    // const size_t shmem_size = (Bc*(d+1) + Br*d + Br*Bc + 5*Br) * sizeof(float);
    dim3 grid(N / BN);
    dim3 block((BN * BN) / (TN * TN));
    const size_t shmem_size = (BN * BN * mul + BN * (BN+1) + BN * BN + 6*BN) * sizeof(float);

    // printf("N %d, d %d, BN %d, Bd %d, TN %d, mul %d, shmem %d\n", N, d, BN, Bd, TN, mul, shmem_size);

    hipFuncSetAttribute(reinterpret_cast<const void*>(s2mm_kernel<BN, TN, mul>), hipFuncAttributeMaxDynamicSharedMemorySize, shmem_size);

    s2mm_kernel<BN, TN, mul><<<grid, block, shmem_size>>>(N, d, Q_d, K_d, V_d, O_d, m_d);
    // cudaDeviceSynchronize();

#if 0
    {
        float* m_h = (float*) malloc(N*sizeof(float));
        cudaMemcpy(m_h, m_d, N*sizeof(float), cudaMemcpyDeviceToHost);
        {
            printf("(N,d,BN,TN)=(%d,%d,%d,%d), ms:\n    ", N,d,BN,TN);
            for(int q=0; q<BN; q++) {
                printf(", %f", m_h[q]);
            }
            printf("\n");
        }

        free(m_h);
    }
#endif

    return 0;
}

int main(int argc, char **argv)
{
        
    struct timeval ts, te;
    size_t cnt;
    float *Q, *K, *V, *O, dur;
    int N, d;

    if (argc != 3) {
        fprintf(stderr, "Usage:\n");
        fprintf(stderr, "  %s d N Compute with matrices filled with ones\n", argv[0]);
        fprintf(stderr, "  %s N -io  Read matrices from stdin and write O to stdout\n", argv[0]);
        return EXIT_FAILURE;
    }

    bool io_arrays = false;
    // if (argc == 3 && strcmp(argv[2], "-io") == 0) {
    //     io_arrays = true;
    // }

    if (io_arrays) {
        N = atoi(argv[1]);
        scanf("%d %d", &d, &N);
    } else {
        d = atoi(argv[1]);
        N = atoi(argv[2]);
    }

    if (N % d != 0) {
        fprintf(stderr, "d must divide N\n");
        return EXIT_FAILURE;
    }

    cnt = N * d;

    fprintf(stderr, "Initializing data...");

    if ((Q = (float*)calloc(cnt, sizeof(float))) == NULL)
        goto mem_failure;

    if ((K = (float*)calloc(cnt, sizeof(float))) == NULL)
        goto mem_failure;

    if ((V = (float*)calloc(cnt, sizeof(float))) == NULL)
        goto mem_failure;

    if ((O = (float*)calloc(cnt, sizeof(float))) == NULL)
        goto mem_failure;

    if (io_arrays) {
        for (int i = 0; i < d * N; i++) scanf("%f", &Q[i]);
        for (int i = 0; i < d * N; i++) scanf("%f", &K[i]);
        for (int i = 0; i < d * N; i++) scanf("%f", &V[i]);
    } else {
        for (size_t i = 0; i < cnt; i++) {
            Q[i] = K[i] = V[i] = 1;
        }
    }

    fprintf(stderr, "done.\n");

    {
        float *Q_d, *K_d, *V_d, *O_d, *m_d, *l_d;

        hipSetDevice(1);

        // allocate memory on device
        hipMalloc((void**) &Q_d, cnt*sizeof(float));
        hipMalloc((void**) &O_d, cnt*sizeof(float));
        hipMalloc((void**) &V_d, cnt*sizeof(float));
        hipMalloc((void**) &K_d, cnt*sizeof(float));
        hipMalloc((void**) &m_d, N*sizeof(float));
        hipMalloc((void**) &l_d, N*sizeof(float));

        // copy host memory to device
        hipMemcpy(Q_d, Q, cnt*sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(V_d, V, cnt*sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(K_d, K, cnt*sizeof(float), hipMemcpyHostToDevice);

        fprintf(stderr, "Warming up...");

        s2mm(m_d, l_d, O_d, Q_d, K_d, V_d, N, d);
        hipDeviceSynchronize();
        gpuAssert( hipPeekAtLastError() );

        fprintf(stderr, "done.\n");
        fprintf(stderr, "Running s2mm...");

        gettimeofday(&ts, NULL);

        for(int i=0; i<GPU_RUNS; i++) {
            s2mm(m_d, l_d, O_d, Q_d, K_d, V_d, N, d);
        }
        hipDeviceSynchronize();
        gpuAssert( hipPeekAtLastError() );

        gettimeofday(&te, NULL);

        fprintf(stderr, "done.\n");

        dur = (double)(te.tv_usec - ts.tv_usec) / 1e6 +
                          (double)(te.tv_sec - ts.tv_sec);
        dur = dur / GPU_RUNS;

        hipMemcpy(O, O_d, cnt*sizeof(float), hipMemcpyDeviceToHost);
        hipDeviceSynchronize();

        if (io_arrays) {
            for (int i = 0; i < N; i++) {
                for (int j = 0; j < d; j++) {
                    if (j > 0) putchar(' ');
                    printf("%f", O[d * i + j]);
                }
                putchar('\n');
            }
        } else {
            // output should be matrix with all elements set to cnt (N * d)
            fprintf(stderr, "L2 norm is %lf (should be %lf)\n", L2(O, cnt), sqrt(cnt));
        }

        /* QK^t is 2N^2d flops, so is PV. softmax(S) (row-wise)
         * exp(S[i]) / sum_j exp(P[i, j] - max(P[i]))
         * is N * (N + 4N) = 5 N^2 flops, but exp is more expensive. */
        fprintf(stderr,
                "Compute rate: %lf Gflops/s, runtime: %lf\n",
                4.0 * d * N * N / dur / 1e9,  dur*1e6);

        hipFree(Q_d);
        hipFree(K_d);
        hipFree(V_d);
        hipFree(O_d);
        hipFree(m_d);
        hipFree(l_d);
    }

    free(Q);
    free(K);
    free(V);
    free(O);

    return EXIT_SUCCESS;

mem_failure:
        printf("Failed to allocate memory.\n");
        return EXIT_FAILURE;
}
